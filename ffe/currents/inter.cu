#include "hip/hip_runtime.h"


#include "../../em-fields/tile.h"
#include "../../tools/mesh.h"
#include "../../tools/signum.h"
#include "rffe.h"

#include <cmath>
#include <iostream>

template<typename F, typename... Args>
__global__ void
  interateXYZKernVari(F fun, int xMax, int yMax, int zMax, Args... args)
{
  //
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int idz = blockIdx.z * blockDim.z + threadIdx.z;

  if(idx >= xMax) return;
  if(idy >= yMax) return;
  if(idz >= zMax) return;

  fun(idx, idy, idz, args...);
}


template<typename F, typename... Args>
__global__ void
  interateXYZKernVariFlat(F fun, int xMax, int yMax, int zMax, Args... args)
{
  //
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i >= xMax * yMax * zMax) return;

  int idx = i % xMax;
  int idy = (i / xMax) % yMax;
  int idz = i / (xMax * yMax);

  fun(idz, idy, idx, args...);
}


void
  interpolateDevEntry(
    toolbox::Mesh<real_short, 3> &f,
    toolbox::Mesh<real_short, 0> &fi,
    const std::array<int, 3> &in,
    const std::array<int, 3> &out)
{

  int im = in[2] == out[2] ? 0 : -out[2];
  int ip = in[2] == out[2] ? 0 : 1 - out[2];

  int jm = in[1] == out[1] ? 0 : -out[1];
  int jp = in[1] == out[1] ? 0 : 1 - out[1];

  int km = in[0] == out[0] ? 0 : -out[0];
  int kp = in[0] == out[0] ? 0 : 1 - out[0];

  dim3 block = { 4, 4, 4 };
  dim3 grid  = { 1 + (f.Nx / 4), 1 + (f.Ny / 4), 1 + (f.Nz / 4) };


  hipHostRegister(&f, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&fi, sizeof(toolbox::Mesh<real_short, 0>), hipHostRegisterMapped);

  toolbox::Mesh<real_short, 3> *f_dev;
  toolbox::Mesh<real_short, 0> *fi_dev;

  hipHostGetDevicePointer(&f_dev, &f, 0);
  hipHostGetDevicePointer(&fi_dev, &fi, 0);

  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(
      int i,
      int j,
      int k,
      toolbox::Mesh<real_short, 3> *f_in,
      toolbox::Mesh<real_short, 0> *fi_in) {
      //
      real_short f11, f10, f01, f00, f1, f0;

      f11 = (*f_in)(i + ip, j + jp, k + km) + (*f_in)(i + ip, j + jp, k + kp);
      f10 = (*f_in)(i + ip, j + jm, k + km) + (*f_in)(i + ip, j + jm, k + kp);
      f01 = (*f_in)(i + im, j + jp, k + km) + (*f_in)(i + im, j + jp, k + kp);
      f00 = (*f_in)(i + im, j + jm, k + km) + (*f_in)(i + im, j + jm, k + kp);
      f1  = f11 + f10;
      f0  = f01 + f00;

      (*fi_in)(i, j, k) = 0.125f * (f1 + f0);
    },
    f.Nx,
    f.Ny,
    f.Nz,
    f_dev,
    fi_dev);

  /*
  auto err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
     fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(err));
  }
  */
}


void
  push_ebDevEntry(ffe::Tile<3> &tile)
{
  //
  // refs to storages
  fields::YeeLattice &m     = tile.get_yee();
  ffe::SkinnyYeeLattice &dm = tile.dF;

  // refs to fields for easier access
  /*
  auto& ex  = m.ex;
  auto& ey  = m.ey;
  auto& ez  = m.ez;

  auto& bx  = m.bx;
  auto& by  = m.by;
  auto& bz  = m.bz;
*/
  real_short c = tile.cfl;

  // dt / dx
  real_short cx = c;
  real_short cy = c;
  real_short cz = c;

  dim3 block = { 4, 4, 4 };
  dim3 grid  = { 1 + (static_cast<int>(tile.mesh_lengths[2]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[1]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[0]) / 4) };

  hipHostRegister(&dm, sizeof(ffe::SkinnyYeeLattice), hipHostRegisterMapped);
  hipHostRegister(&m, sizeof(fields::YeeLattice), hipHostRegisterMapped);

  ffe::SkinnyYeeLattice *dm_dev;
  fields::YeeLattice *m_dev;
  hipHostGetDevicePointer(&dm_dev, &dm, 0);
  hipHostGetDevicePointer(&m_dev, &m, 0);

  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(
      int i,
      int j,
      int k,
      ffe::SkinnyYeeLattice *dm_in,
      fields::YeeLattice *m_in) {
      //
      // dB = dt*curl E
      dm_in->bx(i, j, k) = cz * (m_in->ey(i, j, k + 1) - m_in->ey(i, j, k)) -
                           cy * (m_in->ez(i, j + 1, k) - m_in->ez(i, j, k));
      dm_in->by(i, j, k) = cx * (m_in->ez(i + 1, j, k) - m_in->ez(i, j, k)) -
                           cz * (m_in->ex(i, j, k + 1) - m_in->ex(i, j, k));
      dm_in->bz(i, j, k) = cy * (m_in->ex(i, j + 1, k) - m_in->ex(i, j, k)) -
                           cx * (m_in->ey(i + 1, j, k) - m_in->ey(i, j, k));

      // dE = dt*curl B
      dm_in->ex(i, j, k) = cz * (m_in->by(i, j, k - 1) - m_in->by(i, j, k)) -
                           cy * (m_in->bz(i, j - 1, k) - m_in->bz(i, j, k));
      dm_in->ey(i, j, k) = cx * (m_in->bz(i - 1, j, k) - m_in->bz(i, j, k)) -
                           cz * (m_in->bx(i, j, k - 1) - m_in->bx(i, j, k));
      dm_in->ez(i, j, k) = cy * (m_in->bx(i, j - 1, k) - m_in->bx(i, j, k)) -
                           cx * (m_in->by(i - 1, j, k) - m_in->by(i, j, k));
    },
    static_cast<int>(tile.mesh_lengths[2]),
    static_cast<int>(tile.mesh_lengths[1]),
    static_cast<int>(tile.mesh_lengths[0]),
    dm_dev,
    m_dev);

  /*
  auto err = hipDeviceSynchronize();
if (err != hipSuccess)
{
 fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(err));
}
*/
}


template<>
void
  ffe::rFFE2<3>::add_jperpXDevEntry(ffe::Tile<3> &tile)
{
  fields::YeeLattice &m     = tile.get_yee();
  ffe::SkinnyYeeLattice &dm = tile.dF;

  auto &jx = m.jx;
  // auto& jy  = m.jy;
  // auto& jz  = m.jz;

  real_short dt = tile.cfl;

  dim3 block = { 4, 4, 4 };
  dim3 grid  = { 1 + (static_cast<int>(tile.mesh_lengths[2]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[1]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[0]) / 4) };

  hipHostRegister(&dm, sizeof(ffe::SkinnyYeeLattice), hipHostRegisterMapped);
  hipHostRegister(&jx, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);

  hipHostRegister(&bxf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&byf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&bzf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&exf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&eyf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&ezf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&rhf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);

  ffe::SkinnyYeeLattice *dm_dev;
  toolbox::Mesh<real_short, 3> *jx_dev;

  toolbox::Mesh<real_short, 0> *bxf_dev;
  toolbox::Mesh<real_short, 0> *byf_dev;
  toolbox::Mesh<real_short, 0> *bzf_dev;
  toolbox::Mesh<real_short, 0> *exf_dev;
  toolbox::Mesh<real_short, 0> *eyf_dev;
  toolbox::Mesh<real_short, 0> *ezf_dev;
  toolbox::Mesh<real_short, 0> *rhf_dev;


  hipHostGetDevicePointer(&dm_dev, &dm, 0);
  hipHostGetDevicePointer(&jx_dev, &jx, 0);

  hipHostGetDevicePointer(&bxf_dev, &bxf, 0);
  hipHostGetDevicePointer(&byf_dev, &byf, 0);
  hipHostGetDevicePointer(&bzf_dev, &bzf, 0);
  hipHostGetDevicePointer(&exf_dev, &exf, 0);
  hipHostGetDevicePointer(&eyf_dev, &eyf, 0);
  hipHostGetDevicePointer(&ezf_dev, &ezf, 0);
  hipHostGetDevicePointer(&rhf_dev, &rhf, 0);

  // https://developer.nvidia.com/blog/new-compiler-features-cuda-8/
  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(
      int i,
      int j,
      int k,
      ffe::SkinnyYeeLattice *dm_in,
      toolbox::Mesh<real_short, 3> *jx_in,
      toolbox::Mesh<real_short, 0> *bxf_in,
      toolbox::Mesh<real_short, 0> *byf_in,
      toolbox::Mesh<real_short, 0> *bzf_in,
      toolbox::Mesh<real_short, 0> *exf_in,
      toolbox::Mesh<real_short, 0> *eyf_in,
      toolbox::Mesh<real_short, 0> *ezf_in,
      toolbox::Mesh<real_short, 0> *rhf_in) {
      real_short b2, cur;
      b2 =
        ((*bxf_in)(i, j, k) * (*bxf_in)(i, j, k) +
         (*byf_in)(i, j, k) * (*byf_in)(i, j, k) +
         (*bzf_in)(i, j, k) * (*bzf_in)(i, j, k) + EPS);

      cur = (*rhf_in)(i, j, k) *
            ((*eyf_in)(i, j, k) * (*bzf_in)(i, j, k) -
             (*byf_in)(i, j, k) * (*ezf_in)(i, j, k)) /
            b2;
      (*jx_in)(i, j, k) = cur;
      dm_in->ex(i, j, k) -= dt * cur;
    },
    static_cast<int>(tile.mesh_lengths[2]),
    static_cast<int>(tile.mesh_lengths[1]),
    static_cast<int>(tile.mesh_lengths[0]),
    dm_dev,
    jx_dev,
    bxf_dev,
    byf_dev,
    bzf_dev,
    exf_dev,
    eyf_dev,
    ezf_dev,
    rhf_dev);
  /*
    auto err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
       fprintf(stderr,"GPUassert: %s add_jperpXDevEntry %d\n", hipGetErrorString(err));
    }
    */
}

template<>
void
  ffe::rFFE2<3>::add_jperpYDevEntry(ffe::Tile<3> &tile)
{
  fields::YeeLattice &m     = tile.get_yee();
  ffe::SkinnyYeeLattice &dm = tile.dF;

  // auto& jx  = m.jx;
  auto &jy = m.jy;
  // auto& jz  = m.jz;

  real_short dt = tile.cfl;

  dim3 block = { 4, 4, 4 };
  dim3 grid  = { 1 + (static_cast<int>(tile.mesh_lengths[2]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[1]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[0]) / 4) };

  hipHostRegister(&dm, sizeof(ffe::SkinnyYeeLattice), hipHostRegisterMapped);
  hipHostRegister(&jy, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);

  hipHostRegister(&bxf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&byf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&bzf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&exf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&eyf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&ezf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&rhf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);

  ffe::SkinnyYeeLattice *dm_dev;
  toolbox::Mesh<real_short, 3> *jy_dev;

  toolbox::Mesh<real_short, 0> *bxf_dev;
  toolbox::Mesh<real_short, 0> *byf_dev;
  toolbox::Mesh<real_short, 0> *bzf_dev;
  toolbox::Mesh<real_short, 0> *exf_dev;
  toolbox::Mesh<real_short, 0> *eyf_dev;
  toolbox::Mesh<real_short, 0> *ezf_dev;
  toolbox::Mesh<real_short, 0> *rhf_dev;

  hipHostGetDevicePointer(&dm_dev, &dm, 0);
  hipHostGetDevicePointer(&jy_dev, &jy, 0);

  hipHostGetDevicePointer(&bxf_dev, &bxf, 0);
  hipHostGetDevicePointer(&byf_dev, &byf, 0);
  hipHostGetDevicePointer(&bzf_dev, &bzf, 0);
  hipHostGetDevicePointer(&exf_dev, &exf, 0);
  hipHostGetDevicePointer(&eyf_dev, &eyf, 0);
  hipHostGetDevicePointer(&ezf_dev, &ezf, 0);
  hipHostGetDevicePointer(&rhf_dev, &rhf, 0);

  // https://developer.nvidia.com/blog/new-compiler-features-cuda-8/
  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(
      int i,
      int j,
      int k,
      ffe::SkinnyYeeLattice *dm_in,
      toolbox::Mesh<real_short, 3> *jy_in,
      toolbox::Mesh<real_short, 0> *bxf_in,
      toolbox::Mesh<real_short, 0> *byf_in,
      toolbox::Mesh<real_short, 0> *bzf_in,
      toolbox::Mesh<real_short, 0> *exf_in,
      toolbox::Mesh<real_short, 0> *eyf_in,
      toolbox::Mesh<real_short, 0> *ezf_in,
      toolbox::Mesh<real_short, 0> *rhf_in) {
      real_short b2, cur;
      b2 =
        ((*bxf_in)(i, j, k) * (*bxf_in)(i, j, k) +
         (*byf_in)(i, j, k) * (*byf_in)(i, j, k) +
         (*bzf_in)(i, j, k) * (*bzf_in)(i, j, k) + EPS);

      cur = (*rhf_in)(i, j, k) *
            ((*ezf_in)(i, j, k) * (*bxf_in)(i, j, k) -
             (*exf_in)(i, j, k) * (*bzf_in)(i, j, k)) /
            b2;
      (*jy_in)(i, j, k) = cur;
      dm_in->ey(i, j, k) -= dt * cur;
    },
    static_cast<int>(tile.mesh_lengths[2]),
    static_cast<int>(tile.mesh_lengths[1]),
    static_cast<int>(tile.mesh_lengths[0]),
    dm_dev,
    jy_dev,
    bxf_dev,
    byf_dev,
    bzf_dev,
    exf_dev,
    eyf_dev,
    ezf_dev,
    rhf_dev);
}

template<>
void
  ffe::rFFE2<3>::add_jperpZDevEntry(ffe::Tile<3> &tile)
{
  fields::YeeLattice &m     = tile.get_yee();
  ffe::SkinnyYeeLattice &dm = tile.dF;

  // auto& jx  = m.jx;
  // auto& jy  = m.jy;
  auto &jz = m.jz;

  real_short dt = tile.cfl;

  dim3 block = { 4, 4, 4 };
  dim3 grid  = { 1 + (static_cast<int>(tile.mesh_lengths[2]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[1]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[0]) / 4) };

  hipHostRegister(&dm, sizeof(ffe::SkinnyYeeLattice), hipHostRegisterMapped);
  hipHostRegister(&jz, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);

  hipHostRegister(&bxf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&byf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&bzf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&exf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&eyf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&ezf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&rhf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);

  ffe::SkinnyYeeLattice *dm_dev;
  toolbox::Mesh<real_short, 3> *jz_dev;

  toolbox::Mesh<real_short, 0> *bxf_dev;
  toolbox::Mesh<real_short, 0> *byf_dev;
  toolbox::Mesh<real_short, 0> *bzf_dev;
  toolbox::Mesh<real_short, 0> *exf_dev;
  toolbox::Mesh<real_short, 0> *eyf_dev;
  toolbox::Mesh<real_short, 0> *ezf_dev;
  toolbox::Mesh<real_short, 0> *rhf_dev;

  hipHostGetDevicePointer(&dm_dev, &dm, 0);
  hipHostGetDevicePointer(&jz_dev, &jz, 0);

  hipHostGetDevicePointer(&bxf_dev, &bxf, 0);
  hipHostGetDevicePointer(&byf_dev, &byf, 0);
  hipHostGetDevicePointer(&bzf_dev, &bzf, 0);
  hipHostGetDevicePointer(&exf_dev, &exf, 0);
  hipHostGetDevicePointer(&eyf_dev, &eyf, 0);
  hipHostGetDevicePointer(&ezf_dev, &ezf, 0);
  hipHostGetDevicePointer(&rhf_dev, &rhf, 0);

  // https://developer.nvidia.com/blog/new-compiler-features-cuda-8/
  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(
      int i,
      int j,
      int k,
      ffe::SkinnyYeeLattice *dm_in,
      toolbox::Mesh<real_short, 3> *jz_in,
      toolbox::Mesh<real_short, 0> *bxf_in,
      toolbox::Mesh<real_short, 0> *byf_in,
      toolbox::Mesh<real_short, 0> *bzf_in,
      toolbox::Mesh<real_short, 0> *exf_in,
      toolbox::Mesh<real_short, 0> *eyf_in,
      toolbox::Mesh<real_short, 0> *ezf_in,
      toolbox::Mesh<real_short, 0> *rhf_in) {
      real_short b2, cur;
      b2 =
        ((*bxf_in)(i, j, k) * (*bxf_in)(i, j, k) +
         (*byf_in)(i, j, k) * (*byf_in)(i, j, k) +
         (*bzf_in)(i, j, k) * (*bzf_in)(i, j, k) + EPS);

      cur = (*rhf_in)(i, j, k) *
            ((*exf_in)(i, j, k) * (*byf_in)(i, j, k) -
             (*bxf_in)(i, j, k) * (*eyf_in)(i, j, k)) /
            b2;
      (*jz_in)(i, j, k) = cur;
      dm_in->ez(i, j, k) -= dt * cur;
    },
    static_cast<int>(tile.mesh_lengths[2]),
    static_cast<int>(tile.mesh_lengths[1]),
    static_cast<int>(tile.mesh_lengths[0]),
    dm_dev,
    jz_dev,
    bxf_dev,
    byf_dev,
    bzf_dev,
    exf_dev,
    eyf_dev,
    ezf_dev,
    rhf_dev);
}


template<>
void
  ffe::rFFE2<3>::remove_jparDevEntry(ffe::Tile<3> &tile)
{
  //


  fields::YeeLattice &m     = tile.get_yee();
  ffe::SkinnyYeeLattice &dm = tile.dF;

  real_short cur, b2;
  real_short dt = tile.cfl;

  dim3 block = { 4, 4, 4 };
  dim3 grid  = { 1 + (static_cast<int>(tile.mesh_lengths[2]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[1]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[0]) / 4) };

  hipHostRegister(&dm, sizeof(ffe::SkinnyYeeLattice), hipHostRegisterMapped);
  hipHostRegister(&m, sizeof(fields::YeeLattice), hipHostRegisterMapped);

  hipHostRegister(&bxf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&byf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&bzf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&exf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&eyf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&ezf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);

  fields::YeeLattice *m_dev;
  ffe::SkinnyYeeLattice *dm_dev;

  toolbox::Mesh<real_short, 0> *bxf_dev;
  toolbox::Mesh<real_short, 0> *byf_dev;
  toolbox::Mesh<real_short, 0> *bzf_dev;
  toolbox::Mesh<real_short, 0> *exf_dev;
  toolbox::Mesh<real_short, 0> *eyf_dev;
  toolbox::Mesh<real_short, 0> *ezf_dev;

  hipHostGetDevicePointer(&m_dev, &m, 0);
  hipHostGetDevicePointer(&dm_dev, &dm, 0);

  hipHostGetDevicePointer(&bxf_dev, &bxf, 0);
  hipHostGetDevicePointer(&byf_dev, &byf, 0);
  hipHostGetDevicePointer(&bzf_dev, &bzf, 0);
  hipHostGetDevicePointer(&exf_dev, &exf, 0);
  hipHostGetDevicePointer(&eyf_dev, &eyf, 0);
  hipHostGetDevicePointer(&ezf_dev, &ezf, 0);

  stagger_x_eb(m);
  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(
      int i,
      int j,
      int k,
      fields::YeeLattice *m_in,
      ffe::SkinnyYeeLattice *dm_in,
      toolbox::Mesh<real_short, 0> *bxf_in,
      toolbox::Mesh<real_short, 0> *byf_in,
      toolbox::Mesh<real_short, 0> *bzf_in,
      toolbox::Mesh<real_short, 0> *exf_in,
      toolbox::Mesh<real_short, 0> *eyf_in,
      toolbox::Mesh<real_short, 0> *ezf_in) {
      real_short b2 =
        ((*bxf_in)(i, j, k) * (*bxf_in)(i, j, k) +
         (*byf_in)(i, j, k) * (*byf_in)(i, j, k) +
         (*bzf_in)(i, j, k) * (*bzf_in)(i, j, k) + EPS);
      real_short cur = ((*exf_in)(i, j, k) * (*bxf_in)(i, j, k) +
                        (*eyf_in)(i, j, k) * (*byf_in)(i, j, k) +
                        (*ezf_in)(i, j, k) * (*bzf_in)(i, j, k)) *
                       (*bxf_in)(i, j, k) / b2 / dt;

      m_in->jx(i, j, k) += cur;
      dm_in->ex(i, j, k) = m_in->ex(i, j, k) - cur * dt;
    },
    static_cast<int>(tile.mesh_lengths[2]),
    static_cast<int>(tile.mesh_lengths[1]),
    static_cast<int>(tile.mesh_lengths[0]),
    m_dev,
    dm_dev,
    bxf_dev,
    byf_dev,
    bzf_dev,
    exf_dev,
    eyf_dev,
    ezf_dev);

  stagger_y_eb(m);

  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(
      int i,
      int j,
      int k,
      fields::YeeLattice *m_in,
      ffe::SkinnyYeeLattice *dm_in,
      toolbox::Mesh<real_short, 0> *bxf_in,
      toolbox::Mesh<real_short, 0> *byf_in,
      toolbox::Mesh<real_short, 0> *bzf_in,
      toolbox::Mesh<real_short, 0> *exf_in,
      toolbox::Mesh<real_short, 0> *eyf_in,
      toolbox::Mesh<real_short, 0> *ezf_in) {
      real_short b2 =
        ((*bxf_in)(i, j, k) * (*bxf_in)(i, j, k) +
         (*byf_in)(i, j, k) * (*byf_in)(i, j, k) +
         (*bzf_in)(i, j, k) * (*bzf_in)(i, j, k) + EPS);
      real_short cur = ((*exf_in)(i, j, k) * (*bxf_in)(i, j, k) +
                        (*eyf_in)(i, j, k) * (*byf_in)(i, j, k) +
                        (*ezf_in)(i, j, k) * (*bzf_in)(i, j, k)) *
                       (*byf_in)(i, j, k) / b2 / dt;

      m_in->jy(i, j, k) += cur;
      dm_in->ey(i, j, k) = m_in->ey(i, j, k) - cur * dt;
    },
    static_cast<int>(tile.mesh_lengths[2]),
    static_cast<int>(tile.mesh_lengths[1]),
    static_cast<int>(tile.mesh_lengths[0]),
    m_dev,
    dm_dev,
    bxf_dev,
    byf_dev,
    bzf_dev,
    exf_dev,
    eyf_dev,
    ezf_dev);

  stagger_z_eb(m);
  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(
      int i,
      int j,
      int k,
      fields::YeeLattice *m_in,
      ffe::SkinnyYeeLattice *dm_in,
      toolbox::Mesh<real_short, 0> *bxf_in,
      toolbox::Mesh<real_short, 0> *byf_in,
      toolbox::Mesh<real_short, 0> *bzf_in,
      toolbox::Mesh<real_short, 0> *exf_in,
      toolbox::Mesh<real_short, 0> *eyf_in,
      toolbox::Mesh<real_short, 0> *ezf_in) {
      real_short b2 =
        ((*bxf_in)(i, j, k) * (*bxf_in)(i, j, k) +
         (*byf_in)(i, j, k) * (*byf_in)(i, j, k) +
         (*bzf_in)(i, j, k) * (*bzf_in)(i, j, k) + EPS);
      real_short cur = ((*exf_in)(i, j, k) * (*bxf_in)(i, j, k) +
                        (*eyf_in)(i, j, k) * (*byf_in)(i, j, k) +
                        (*ezf_in)(i, j, k) * (*bzf_in)(i, j, k)) *
                       (*bzf_in)(i, j, k) / b2 / dt;

      m_in->jz(i, j, k) += cur;
      dm_in->ez(i, j, k) = m_in->ez(i, j, k) - cur * dt;
    },
    static_cast<int>(tile.mesh_lengths[2]),
    static_cast<int>(tile.mesh_lengths[1]),
    static_cast<int>(tile.mesh_lengths[0]),
    m_dev,
    dm_dev,
    bxf_dev,
    byf_dev,
    bzf_dev,
    exf_dev,
    eyf_dev,
    ezf_dev);

}


template<>
void
  ffe::rFFE2<3>::limit_eDevEntry(ffe::Tile<3> &tile)
{

  fields::YeeLattice &m     = tile.get_yee();
  ffe::SkinnyYeeLattice &dm = tile.dF;

  real_short dt = tile.cfl;
  real_short e2, b2, diss, cur;


  hipHostRegister(&dm, sizeof(ffe::SkinnyYeeLattice), hipHostRegisterMapped);
  hipHostRegister(&m, sizeof(fields::YeeLattice), hipHostRegisterMapped);

  hipHostRegister(&bxf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&byf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&bzf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&exf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&eyf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);
  hipHostRegister(&ezf, sizeof(toolbox::Mesh<real_short, 3>), hipHostRegisterMapped);

  fields::YeeLattice *m_dev;
  ffe::SkinnyYeeLattice *dm_dev;

  toolbox::Mesh<real_short, 0> *bxf_dev;
  toolbox::Mesh<real_short, 0> *byf_dev;
  toolbox::Mesh<real_short, 0> *bzf_dev;
  toolbox::Mesh<real_short, 0> *exf_dev;
  toolbox::Mesh<real_short, 0> *eyf_dev;
  toolbox::Mesh<real_short, 0> *ezf_dev;

  hipHostGetDevicePointer(&m_dev, &m, 0);
  hipHostGetDevicePointer(&dm_dev, &dm, 0);

  hipHostGetDevicePointer(&bxf_dev, &bxf, 0);
  hipHostGetDevicePointer(&byf_dev, &byf, 0);
  hipHostGetDevicePointer(&bzf_dev, &bzf, 0);
  hipHostGetDevicePointer(&exf_dev, &exf, 0);
  hipHostGetDevicePointer(&eyf_dev, &eyf, 0);
  hipHostGetDevicePointer(&ezf_dev, &ezf, 0);

  /*
  dim3 block = { 256,1,1 };
  dim3 grid  = { 1 +
               (static_cast<int>(
                  tile.mesh_lengths[2] * tile.mesh_lengths[1] * tile.mesh_lengths[0]) /
                256) ,1,1};
*/

dim3 block = { 4, 4, 4 };
dim3 grid  = { 1 + (static_cast<int>(tile.mesh_lengths[2]) / 4),
              1 + (static_cast<int>(tile.mesh_lengths[1]) / 4),
              1 + (static_cast<int>(tile.mesh_lengths[0]) / 4) };

  stagger_x_eb(m);
  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(
      int i,
      int j,
      int k,
      fields::YeeLattice *m_in,
      ffe::SkinnyYeeLattice *dm_in,
      toolbox::Mesh<real_short, 0> *bxf_in,
      toolbox::Mesh<real_short, 0> *byf_in,
      toolbox::Mesh<real_short, 0> *bzf_in,
      toolbox::Mesh<real_short, 0> *exf_in,
      toolbox::Mesh<real_short, 0> *eyf_in,
      toolbox::Mesh<real_short, 0> *ezf_in) {
      real_short e2 = (*exf_in)(i, j, k) * (*exf_in)(i, j, k) +
                      (*eyf_in)(i, j, k) * (*eyf_in)(i, j, k) +
                      (*ezf_in)(i, j, k) * (*ezf_in)(i, j, k) + EPS;
      real_short b2 = (*bxf_in)(i, j, k) * (*bxf_in)(i, j, k) +
                      (*byf_in)(i, j, k) * (*byf_in)(i, j, k) +
                      (*bzf_in)(i, j, k) * (*bzf_in)(i, j, k) + EPS;

      real_short diss = 1.0;
      if(e2 > b2) diss = std::sqrt(b2 / e2);

      real_short cur = (1. - diss) * dm_in->ex(i, j, k) / dt;
      m_in->jx(i, j, k) += cur;
      m_in->ex(i, j, k) = diss * dm_in->ex(i, j, k);
    },
    static_cast<int>(tile.mesh_lengths[2]),
    static_cast<int>(tile.mesh_lengths[1]),
    static_cast<int>(tile.mesh_lengths[0]),
    m_dev,
    dm_dev,
    bxf_dev,
    byf_dev,
    bzf_dev,
    exf_dev,
    eyf_dev,
    ezf_dev);

  stagger_y_eb(m);
  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(
      int i,
      int j,
      int k,
      fields::YeeLattice *m_in,
      ffe::SkinnyYeeLattice *dm_in,
      toolbox::Mesh<real_short, 0> *bxf_in,
      toolbox::Mesh<real_short, 0> *byf_in,
      toolbox::Mesh<real_short, 0> *bzf_in,
      toolbox::Mesh<real_short, 0> *exf_in,
      toolbox::Mesh<real_short, 0> *eyf_in,
      toolbox::Mesh<real_short, 0> *ezf_in) {
      real_short e2 = (*exf_in)(i, j, k) * (*exf_in)(i, j, k) +
                      (*eyf_in)(i, j, k) * (*eyf_in)(i, j, k) +
                      (*ezf_in)(i, j, k) * (*ezf_in)(i, j, k) + EPS;
      real_short b2 = (*bxf_in)(i, j, k) * (*bxf_in)(i, j, k) +
                      (*byf_in)(i, j, k) * (*byf_in)(i, j, k) +
                      (*bzf_in)(i, j, k) * (*bzf_in)(i, j, k) + EPS;

      real_short diss = 1.0;
      if(e2 > b2) diss = std::sqrt(b2 / e2);

      real_short cur = (1. - diss) * dm_in->ey(i, j, k) / dt;
      m_in->jy(i, j, k) += cur;
      m_in->ey(i, j, k) = diss * dm_in->ey(i, j, k);
    },
    static_cast<int>(tile.mesh_lengths[2]),
    static_cast<int>(tile.mesh_lengths[1]),
    static_cast<int>(tile.mesh_lengths[0]),
    m_dev,
    dm_dev,
    bxf_dev,
    byf_dev,
    bzf_dev,
    exf_dev,
    eyf_dev,
    ezf_dev);

  stagger_z_eb(m);
  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(
      int i,
      int j,
      int k,
      fields::YeeLattice *m_in,
      ffe::SkinnyYeeLattice *dm_in,
      toolbox::Mesh<real_short, 0> *bxf_in,
      toolbox::Mesh<real_short, 0> *byf_in,
      toolbox::Mesh<real_short, 0> *bzf_in,
      toolbox::Mesh<real_short, 0> *exf_in,
      toolbox::Mesh<real_short, 0> *eyf_in,
      toolbox::Mesh<real_short, 0> *ezf_in) {
      real_short e2 = (*exf_in)(i, j, k) * (*exf_in)(i, j, k) +
                      (*eyf_in)(i, j, k) * (*eyf_in)(i, j, k) +
                      (*ezf_in)(i, j, k) * (*ezf_in)(i, j, k) + EPS;
      real_short b2 = (*bxf_in)(i, j, k) * (*bxf_in)(i, j, k) +
                      (*byf_in)(i, j, k) * (*byf_in)(i, j, k) +
                      (*bzf_in)(i, j, k) * (*bzf_in)(i, j, k) + EPS;

      real_short diss = 1.0;
      if(e2 > b2) diss = std::sqrt(b2 / e2);

      real_short cur = (1. - diss) * dm_in->ez(i, j, k) / dt;
      m_in->jz(i, j, k) += cur;
      m_in->ez(i, j, k) = diss * dm_in->ez(i, j, k);
    },
    static_cast<int>(tile.mesh_lengths[2]),
    static_cast<int>(tile.mesh_lengths[1]),
    static_cast<int>(tile.mesh_lengths[0]),
    m_dev,
    dm_dev,
    bxf_dev,
    byf_dev,
    bzf_dev,
    exf_dev,
    eyf_dev,
    ezf_dev);
}


template<>
void
  ffe::rFFE2<3>::update_ebDevEntry(
    ffe::Tile<3> &tile,
    real_short c1,
    real_short c2,
    real_short c3)
{

  fields::YeeLattice &m     = tile.get_yee();
  ffe::SkinnyYeeLattice &n  = tile.Fn;
  ffe::SkinnyYeeLattice &dm = tile.dF;
  // real_short dt = tile.cfl;


  hipHostRegister(&dm, sizeof(ffe::SkinnyYeeLattice), hipHostRegisterMapped);
  hipHostRegister(&n, sizeof(ffe::SkinnyYeeLattice), hipHostRegisterMapped);
  hipHostRegister(&m, sizeof(fields::YeeLattice), hipHostRegisterMapped);

  fields::YeeLattice *m_dev;
  ffe::SkinnyYeeLattice *dm_dev;
  ffe::SkinnyYeeLattice *n_dev;

  hipHostGetDevicePointer(&m_dev, &m, 0);
  hipHostGetDevicePointer(&dm_dev, &dm, 0);
  hipHostGetDevicePointer(&n_dev, &n, 0);

  dim3 block = { 4, 4, 4 };
  dim3 grid  = { 1 + (static_cast<int>(tile.mesh_lengths[2]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[1]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[0]) / 4) };


  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(
      int i,
      int j,
      int k,
      fields::YeeLattice *m_in,
      ffe::SkinnyYeeLattice *dm_in,
      ffe::SkinnyYeeLattice *n_in) {
      // RK3 E update
      m_in->ex(i, j, k) =
        c1 * n_in->ex(i, j, k) + c2 * m_in->ex(i, j, k) + c3 * dm_in->ex(i, j, k);
      m_in->ey(i, j, k) =
        c1 * n_in->ey(i, j, k) + c2 * m_in->ey(i, j, k) + c3 * dm_in->ey(i, j, k);
      m_in->ez(i, j, k) =
        c1 * n_in->ez(i, j, k) + c2 * m_in->ez(i, j, k) + c3 * dm_in->ez(i, j, k);

      // RK3 B update
      m_in->bx(i, j, k) =
        c1 * n_in->bx(i, j, k) + c2 * m_in->bx(i, j, k) + c3 * dm_in->bx(i, j, k);
      m_in->by(i, j, k) =
        c1 * n_in->by(i, j, k) + c2 * m_in->by(i, j, k) + c3 * dm_in->by(i, j, k);
      m_in->bz(i, j, k) =
        c1 * n_in->bz(i, j, k) + c2 * m_in->bz(i, j, k) + c3 * dm_in->bz(i, j, k);

      // variable switch for 1) e > b and 2) j_par calcs.
      // Enables to calculate both of the above as independent
      // corrections because interpolation is done via m.ex
      // meshes and results are stored in dm.ex meshes:
      dm_in->ex(i, j, k) = m_in->ex(i, j, k);
      dm_in->ey(i, j, k) = m_in->ey(i, j, k);
      dm_in->ez(i, j, k) = m_in->ez(i, j, k);
    },
    static_cast<int>(tile.mesh_lengths[2]),
    static_cast<int>(tile.mesh_lengths[1]),
    static_cast<int>(tile.mesh_lengths[0]),
    m_dev,
    dm_dev,
    n_dev);

}

template<>
void
  ffe::rFFE2<3>::copy_ebDevEntry(ffe::Tile<3> &tile)
{

  fields::YeeLattice &m    = tile.get_yee();
  ffe::SkinnyYeeLattice &n = tile.Fn;

  hipHostRegister(&n, sizeof(ffe::SkinnyYeeLattice), hipHostRegisterMapped);
  hipHostRegister(&m, sizeof(fields::YeeLattice), hipHostRegisterMapped);

  fields::YeeLattice *m_dev;
  ffe::SkinnyYeeLattice *n_dev;

  hipHostGetDevicePointer(&m_dev, &m, 0);
  hipHostGetDevicePointer(&n_dev, &n, 0);

  dim3 block = { 4, 4, 4 };
  dim3 grid  = { 1 + (static_cast<int>(tile.mesh_lengths[2]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[1]) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[0]) / 4) };

  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(
      int i,
      int j,
      int k,
      fields::YeeLattice *m_in,
      ffe::SkinnyYeeLattice *n_in) {
      n_in->ex(i, j, k) = m_in->ex(i, j, k);
      n_in->ey(i, j, k) = m_in->ey(i, j, k);
      n_in->ez(i, j, k) = m_in->ez(i, j, k);

      n_in->bx(i, j, k) = m_in->bx(i, j, k);
      n_in->by(i, j, k) = m_in->by(i, j, k);
      n_in->bz(i, j, k) = m_in->bz(i, j, k);
    },
    static_cast<int>(tile.mesh_lengths[2]),
    static_cast<int>(tile.mesh_lengths[1]),
    static_cast<int>(tile.mesh_lengths[0]),
    m_dev,
    n_dev);
}


template<>
void
  ffe::rFFE2<3>::comp_rhoDevEntry(ffe::Tile<3> &tile)
{
  fields::YeeLattice &mesh = tile.get_yee();

  hipHostRegister(&mesh, sizeof(fields::YeeLattice), hipHostRegisterMapped);
  fields::YeeLattice *m_dev;
  hipHostGetDevicePointer(&m_dev, &mesh, 0);

  dim3 block = { 4, 4, 4 };
  dim3 grid  = { 1 + (static_cast<int>(tile.mesh_lengths[2]+2) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[1]+2) / 4),
                1 + (static_cast<int>(tile.mesh_lengths[0]+2) / 4) };

  // NOTE: compute rho from -1 to +1 because later on re-stagger it
  // and need the guard zones for interpolation
  interateXYZKernVari<<<grid, block>>>(
    [=] __device__ __host__(int i, int j, int k, fields::YeeLattice *m_in) {
        m_in->rho(i-1,j-1,k-1) = 
          (m_in->ex(i-1,j-1,k-1) - m_in->ex(i-1-1,j-1,  k-1  )) +
          (m_in->ey(i-1,j-1,k-1) - m_in->ey(i-1  ,j-1-1,k-1  )) + 
          (m_in->ez(i-1,j-1,k-1) - m_in->ez(i-1  ,j-1,  k-1-1));
    },
    static_cast<int>(tile.mesh_lengths[2])+2,
    static_cast<int>(tile.mesh_lengths[1])+2,
    static_cast<int>(tile.mesh_lengths[0])+2,
    m_dev);
}